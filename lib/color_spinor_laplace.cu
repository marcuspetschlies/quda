#include "hip/hip_runtime.h"
#include <transfer.h>
#include <quda_internal.h>
#include <quda_matrix.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <index_helper.cuh>
#include <color_spinor.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>

namespace quda {

  template <typename Float, int Ns, int Nc, QudaReconstructType gRecon>
  struct LaplaceSmearingArg {
    typedef typename colorspinor_mapper<Float,Ns,Nc>::type F;
    typedef typename gauge_mapper<Float,gRecon>::type G;

    F out;                // output vector field
    const F in;           // input vector field
    const G U;            // the gauge field
    const Float A;        // A parameter
    const Float B;        // B parameter
    const int parity;     // only use this for single parity fields
    const int nParity;    // number of parities we're working on
    const int nFace;      // hard code to 1 for now
    const int dim[5];     // full lattice dimensions
    const int commDim[4]; // whether a given dimension is partitioned or not
    const int volumeCB;   // checkerboarded volume

    LaplaceSmearingArg(ColorSpinorField &out, const ColorSpinorField &in, int parity, const GaugeField &U,
                       Float A, Float B)
      : out(out), in(in), U(U), A(A), B(B), parity(parity), nParity(in.SiteSubset()), nFace(1),
        dim{ (3-nParity) * in.X(0), in.X(1), in.X(2), in.X(3), 1 },
      commDim{comm_dim_partitioned(0), comm_dim_partitioned(1), comm_dim_partitioned(2), comm_dim_partitioned(3)},
      volumeCB(in.VolumeCB())
    {
      if (in.FieldOrder() != QUDA_FLOAT2_FIELD_ORDER || !U.isNative())
        errorQuda("Unsupported field order colorspinor=%d gauge=%d combination\n", in.FieldOrder(), U.FieldOrder());
    }
  };

  /**
     Computes out = sum_mu U_mu(x)in(x+d) + U^\dagger_mu(x-d)in(x-d)
     @param[out] out The out result field
     @param[in] U The gauge field
     @param[in] in The input field
     @param[in] x_cb The checkerboarded site index
     @param[in] parity The site parity
  */
  template <typename Float, int Nc, typename Vector, typename Arg>
  __device__ __host__ inline void computeNeighborSum(Vector &out, Arg &arg, int x_cb, int parity) {

    typedef Matrix<complex<Float>,Nc> Link;
    const int their_spinor_parity = (arg.nParity == 2) ? 1-parity : 0;

    int coord[5];
    getCoords(coord, x_cb, arg.dim, parity);
    coord[4] = 0;

#pragma unroll
    for (int dir=0; dir<4; dir++) { // loop over spatial and temporal directions

      //Forward gather - compute fwd offset for vector fetch
      const int fwd_idx = linkIndexP1(coord, arg.dim, dir);

      if ( arg.commDim[dir] && (coord[dir] + arg.nFace >= arg.dim[dir]) ) {
        const int ghost_idx = ghostFaceIndex<1>(coord, arg.dim, dir, arg.nFace);

        const Link U = arg.U(dir, x_cb, parity);
	const Vector in = arg.in.Ghost(dir, 1, ghost_idx, their_spinor_parity);

        out += U * in;
      } else {
        const Link U = arg.U(dir, x_cb, parity);
	const Vector in = arg.in(fwd_idx, their_spinor_parity);

        out += U * in;
      }

      //Backward gather - compute back offset for spinor and gauge fetch
      const int back_idx = linkIndexM1(coord, arg.dim, dir);
      const int gauge_idx = back_idx;

      if ( arg.commDim[dir] && (coord[dir] - arg.nFace < 0) ) {
        const int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, dir, arg.nFace);

        const Link U = arg.U.Ghost(dir, ghost_idx, 1-parity);
	const Vector in = arg.in.Ghost(dir, 0, ghost_idx, their_spinor_parity);

        out += conj(U) * in;
      } else {
        const Link U = arg.U(dir, gauge_idx, 1-parity);
	const Vector in = arg.in(back_idx, their_spinor_parity);

        out += conj(U) * in;
      }
    }
  }

  //out(x) = A in(x) + B computeNeighborSum(out, x)
  template <typename Float, int Ns, int Nc, typename Arg>
  __device__ __host__ inline void computeLaplaceStep(Arg &arg, int x_cb, int parity)
  {
    typedef ColorSpinor<Float,Nc,Ns> Vector;
    Vector out;

    computeNeighborSum<Float,Nc>(out, arg, x_cb, parity);

    Vector in;
    arg.in.load((Float*)in.data, x_cb, parity);
    out = arg.A*in + arg.B*out;
    
    arg.out(x_cb, parity) = out;
  }

  // CPU kernel for applying a laplace smearing step to a vector
  template <typename Float, int Ns, int Nc, typename Arg>
  void laplaceStepCPU(Arg arg)
  {

    for (int parity= 0; parity < arg.nParity; parity++) {
      // for full fields then set parity from loop else use arg setting
      parity = (arg.nParity == 2) ? parity : arg.parity;

      for (int x_cb = 0; x_cb < arg.volumeCB; x_cb++) { // 4-d volume
        computeLaplaceStep<Float,Ns,Nc>(arg, x_cb, parity);
      } // 4-d volumeCB
    } // parity

  }

  // GPU Kernel for applying a laplace smearing step to a vector
  template <typename Float, int Ns, int Nc, typename Arg>
  __global__ void laplaceStepGPU(Arg arg)
  {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;

    // for full fields set parity from y thread index else use arg setting
    int parity = blockDim.y*blockIdx.y + threadIdx.y;

    if (x_cb >= arg.volumeCB) return;
    if (parity >= arg.nParity) return;
    parity = (arg.nParity == 2) ? parity : arg.parity;

    computeLaplaceStep<Float,Ns,Nc>(arg, x_cb, parity);
  }

  template <typename Float, int Ns, int Nc, typename Arg>
  class LaplaceSmearing : public TunableVectorY {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

    long long flops() const
    {
      return (2*3*Ns*Nc*(8*Nc-2) + 2*3*Nc*Ns )*arg.nParity*(long long)meta.VolumeCB();
    }
    long long bytes() const
    {
      return arg.out.Bytes() + (2*3+1)*arg.in.Bytes() + arg.nParity*2*3*arg.U.Bytes()*meta.VolumeCB();
    }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }
    unsigned int maxBlockSize() const { return deviceProp.maxThreadsPerBlock / arg.nParity; }

  public:
    LaplaceSmearing(Arg &arg, const ColorSpinorField &meta) : TunableVectorY(arg.nParity), arg(arg), meta(meta)
    {
      strcpy(aux, meta.AuxString());
      strcat(aux, comm_dim_partitioned_string());
    }
    virtual ~LaplaceSmearing() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
        laplaceStepCPU<Float,Ns,Nc>(arg);
      } else {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        laplaceStepGPU<Float,Ns,Nc> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  };

  template<typename Float, int Ns, int Nc, QudaReconstructType gRecon>
  void laplaceStep(ColorSpinorField &out, const ColorSpinorField &in, int parity,
		     const GaugeField& U, double A, double B)
  {
    LaplaceSmearingArg<Float,Ns,Nc,gRecon> arg(out, in, parity, U, A, B);
    LaplaceSmearing<Float,Ns,Nc,LaplaceSmearingArg<Float,Ns,Nc,gRecon> > laplace(arg, in);
    laplace.apply(0);
  }

  // template on the gauge reconstruction
  template<typename Float, int Ns, int Nc>
  void laplaceStep(ColorSpinorField &out, const ColorSpinorField &in, int parity,
		     const GaugeField& U, double A, double B)
  {
    if (U.Reconstruct() == QUDA_RECONSTRUCT_NO) {
      laplaceStep<Float,Ns,Nc,QUDA_RECONSTRUCT_NO>(out, in, parity, U, A, B);
    } else if(U.Reconstruct() == QUDA_RECONSTRUCT_12) {
      laplaceStep<Float,Ns,Nc,QUDA_RECONSTRUCT_12>(out, in, parity, U, A, B);
    } else if(U.Reconstruct() == QUDA_RECONSTRUCT_8) {
      laplaceStep<Float,Ns,Nc,QUDA_RECONSTRUCT_8>(out, in, parity, U, A, B);
    } else {
      errorQuda("Reconstruction type %d of origin gauge field not supported", U.Reconstruct());
    }
  }


  // template on the number of colors
  template<typename Float, int Ns>
  void laplaceStep(ColorSpinorField &out, const ColorSpinorField &in, int parity,
		     const GaugeField& U, double A, double B)
  {
    if (out.Ncolor() != in.Ncolor()) {
      errorQuda("Orign and destination fields must have the same number of colors\n");
    }

    if (out.Ncolor() == 3 ) {
      laplaceStep<Float,Ns,3>(out, in, parity, U, A, B);
    } else {
      errorQuda(" is not implemented for Ncolor!=3");
    }
  }

  // template on the number of spins
  template<typename Float>
  void laplaceStep(ColorSpinorField &out, const ColorSpinorField &in, int parity,
		     const GaugeField& U, double A, double B)
  {
    if(out.Nspin() != in.Nspin()) {
      errorQuda("Orign and destination fields must have the same number of spins\n");
    }

    if (out.Nspin() == 4 ){
      laplaceStep<Float,4>(out, in, parity, U, A, B);
    }else if (in.Nspin() == 1 ){
      laplaceStep<Float,1>(out, in, parity, U, A, B);
    }else{
      errorQuda("Nspin %d not supported", out.Nspin());
    }
  }

  /**
     Apply a generic Laplace smearing step
     Computes out(x) = A*in(x)  + B*\sum_mu (U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu))
     @param[out] out The out result field
     @param[in] in The in spinor field
     @param[in] U The gauge field
     @param[in] A The scaling factor for in(x)
     @param[in] B The scaling factor for \sum_mu (U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu))
  */
  // template on the precision
  void laplaceStep(ColorSpinorField &out, const ColorSpinorField &in, int parity,
		     const GaugeField& U, double A, double B)
  {
    if (in.V() == out.V()) {
      errorQuda("Orign and destination fields must be different pointers");
    }

    // check precisions match
    checkPrecision(out, in, U);

    // check all locations match
    checkLocation(out, in, U);

    const int nFace = 1;
    in.exchangeGhost((QudaParity)(1-parity), nFace, 0); // last parameter is dummy

    if (out.Precision() == QUDA_SINGLE_PRECISION){
      laplaceStep<float>(out, in, parity, U, A, B);
    } else if(out.Precision() == QUDA_DOUBLE_PRECISION) {
      laplaceStep<double>(out, in, parity, U, A, B);
    } else {
      errorQuda("Precision %d not supported", out.Precision());
    }

    in.bufferIndex = (1 - in.bufferIndex);
    return;
  }

  /**
     Apply a standard Laplace smearing step
     Computes out(x) = 1/(1+6*alpha)*(in(x)  + alpha*\sum_mu (U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu)))
     @param[out] out The out result field
     @param[in] in The in spinor field
     @param[in] U The gauge field
     @param[in] alpha The smearing parameter
  */
  void laplaceStep(ColorSpinorField &out, const ColorSpinorField &in, int parity, const GaugeField& U, double alpha)
  {
    laplaceStep(out, in, parity, U, 1./(1.+6.*alpha), alpha/(1.+6.*alpha));
  }
} // namespace quda
